#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>



#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}


__global__ void setup_kernel(hiprandState *state){

		  int idx = threadIdx.x+blockDim.x*blockIdx.x;
	  hiprand_init(1234, idx, 0, &state[idx]); 
}

__global__ void generate_random(hiprandState *my_curandstate,float* res,int elems_per_thread){
		  int idx = elems_per_thread*(threadIdx.x+blockDim.x*blockIdx.x);

		for(int j =0; j< elems_per_thread ; j++){
		res[idx+j]=hiprand_uniform(my_curandstate+idx);
		}
}



// multiply accumulate in b;
__global__
void myVeryChrunchyFunction(float* a,float* b, int elems_per_thread){
int i = elems_per_thread*(blockDim.x * blockIdx.x + threadIdx.x);	

	for(int j = 0; j<elems_per_thread; j++){
	int idx = i+j;
		b[idx]=a[idx]*b[idx];
	}

}


int main(void){

	int num_blocks =14;
	int num_threads_per_block=256;
	int elems_per_thread = 360;
	int num_elements = num_blocks*num_threads_per_block*elems_per_thread;

	float *host_array_b =(float *) malloc(num_elements*sizeof(float));
	float *device_array_a;
	float *device_array_b;
	
gpuErrchk(hipMalloc(&device_array_a,num_elements*sizeof(float)));
gpuErrchk(hipMalloc(&device_array_b,num_elements*sizeof(float)));
	   hiprandState *d_state;
gpuErrchk(hipMalloc(&d_state, sizeof(hiprandState)*num_blocks*num_threads_per_block));
	
	

	setup_kernel<<<num_blocks,num_threads_per_block>>>(d_state);
	generate_random<<<num_blocks,num_threads_per_block>>>(d_state,device_array_a,elems_per_thread);
	generate_random<<<num_blocks,num_threads_per_block>>>(d_state,device_array_b,elems_per_thread);
	myVeryChrunchyFunction<<<num_blocks,num_threads_per_block>>>(device_array_a,device_array_b,elems_per_thread);
	
	hipMemcpy(host_array_b,device_array_b,num_elements*sizeof(float),hipMemcpyDeviceToHost);


	time_t t;
	srand((unsigned) time(&t));
	int index = rand() % num_elements;
	printf("Picking random element:\n\tlocation: %d \n\tvalue: %f\n",index,host_array_b[index]);

	hipFree(device_array_a);
	free(host_array_b);
	hipFree(device_array_b);
	hipFree(d_state);

	return 0;

}

