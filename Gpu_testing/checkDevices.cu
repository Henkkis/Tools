#include <stdio.h> 

int main() {
  int nDevices;
  hipGetDeviceCount(&nDevices);
  for (int i = 0; i < nDevices; i++) {
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, i);
    printf("Device Number: %d\n", i);
    printf("  Device name: %s\n", prop.name);
    printf("  Memory Clock Rate (KHz): %d\n",
           prop.memoryClockRate);
    printf("  Memory Bus Width (bits): %d\n",
           prop.memoryBusWidth);
    printf("  Peak Memory Bandwidth (GB/s): %f\n",
           2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);
    printf("  pciBusID %d\n",prop.pciBusID);
    printf("  pciDeviceID %d\n",prop.pciDeviceID);
    printf("  Compute Capability: %d.%d\n",prop.major,prop.minor);
    printf("  totalGlobalMem:%zu\n",prop.totalGlobalMem);
    printf("  warpSize:%d\n",prop.warpSize);
    printf("  regsPerBlock:%d\n",prop.regsPerBlock);
    printf("  sharedMemPerBlock:%d\n",prop.sharedMemPerBlock);
  
  	printf("\n");
  }
}



