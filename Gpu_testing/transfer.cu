#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>


__global__
void add_one(int n, double *array,int updates_per_thread){
int i = blockDim.x * blockIdx.x + threadIdx.x;	



for( unsigned int j = i*updates_per_thread ; j < (i+1)*updates_per_thread; j++){

	if( j < n){
		array[j] += j;
	
	}
}

}
int main(void){
	
	double transfer_memory_size = 1e10;
	int num_elements = (transfer_memory_size/sizeof(double));
	double *host_array = (double*) malloc(sizeof(double)*num_elements); 
	double *device_array;
	hipMalloc(&device_array,num_elements*sizeof(double));
	
	unsigned int i;
	for(  i = 0;  i < num_elements ; i++){
		host_array[i] = (double) i;
	}

	hipMemcpy(device_array,host_array,num_elements*sizeof(double),hipMemcpyHostToDevice);


	int gridsize = 5000;
	if(num_elements < gridsize*256){
		gridsize = (num_elements+255)/256;
	}
	



	int updates_per_thread = (num_elements+gridsize*256)/(gridsize*256);
	printf("Gridsize: %d\nNumber of elements: %d\nSpawned threads: %d\n", gridsize, num_elements,gridsize*256,updates_per_thread);
	add_one<<<gridsize,256>>>(num_elements,device_array,updates_per_thread);
	hipMemcpy(host_array,device_array,num_elements*sizeof(double),hipMemcpyDeviceToHost);


	time_t t;
	srand((unsigned) time(&t));
	int index = rand() % num_elements;
	printf("Picking random element:\n\tlocation: %d \n\tvalue: %f\n",index,host_array[index]);
	printf("Last element: %f\n", host_array[num_elements-1] );


	free(host_array);
	hipFree(device_array);
}
